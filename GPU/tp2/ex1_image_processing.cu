#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include "image.h"

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

template <typename T>
__device__ inline T* get_ptr(T *img, int i, int j, int C, size_t pitch) {
    return reinterpret_cast<T*>(reinterpret_cast<char*>(img) + (i * pitch + j) /*- ((i * pitch + j)%C)*/ * sizeof(T));
}

__global__ void process(int N, int M, int C, int pitch, float* img)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < M && j < N) {
        float* pixel = get_ptr(img,i,j,C,pitch);
        float newColor = 0;
        printf("pixel avec les pointeurs = %f , %f , %f\n", *pixel);
        printf("pixel avec les indices = %f , %f , %f\n", pixel[0]);
        for (int k=0; k<C; k+=1)
        {
            newColor += pixel[k];
        }
        newColor =  newColor/C;
        for (int k=0; k<C; k+=1)
        {
            pixel[k] = newColor;
        }
    }
}

int main(int argc, char const *argv[])
{
    const std::string filename = argc >= 2 ? argv[1] : "image.jpg";
    std::cout << "filename = " << filename << std::endl;
    int M = 0;
    int N = 0;
    int C = 0;
    float* img = image::load(filename, &N, &M, &C);
    std::cout << "N (columns, width) = " << N << std::endl;
    std::cout << "M (rows, height) = " << M << std::endl;
    std::cout << "C (channels, depth) = " << C << std::endl;

    size_t pitch;

    float* cpy;
    CUDA_CHECK(hipMallocPitch(&cpy, &pitch, N * sizeof(float), M));
    CUDA_CHECK(hipMemcpy2D(cpy, pitch, img, N * sizeof(float), N * sizeof(float), M, hipMemcpyHostToDevice));
    
    // launch kernel
    dim3 block_dim(32, 32);
    dim3 grid_dim((M + block_dim.x - 1) / block_dim.x, (N + block_dim.y - 1) / block_dim.y);
    process<<<grid_dim, block_dim>>>(N,M,C,pitch,cpy);
    
    // copy device memory back to host memory
    CUDA_CHECK(hipMemcpy2D(img, pitch, cpy, N * sizeof(float), N * sizeof(float), M, hipMemcpyDeviceToHost));
    image::save("result.jpg", N, M, C, img);

    hipFree(cpy);
    free(img);

    return 0;
}
