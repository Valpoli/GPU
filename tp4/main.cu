#include "hip/hip_runtime.h"
#include "make_vector.h"
#include <iostream>

#define STATIC_SIZE 64

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

std::vector<int> scan_exclu(std::vector<int> table)
{
    std::size_t size = table.size();
    std::vector<int> res = make_vector((int) size);
    res[0] = 0;
    for (std::size_t i = 1; i < size; ++i)
    {
        res[i] = res[i-1] + table[i-1];
    }
    return res;
}

__global__ void scan1(int* x, int N)
{
    __shared__ int buffers[2*N];
    int in = 0;
    int out = N;
    int i = threadIdx.x;
    buffers[i] = x[i];
    __syncthreads();
    if (i == 0)
    {
        x[i] = 0;
    }
    else
    {
        int step = (int) log2(N);
        for (int n = 0; n < step; ++n)
        {
            offset = 2^step ;
            if (offset <= i)
            {
                buffers[i + out] = buffer[i + in] + buffer[i + in - offset];
            }
            else
            {
                buffers[i + out] = buffer[i + in];
            }
            syncthreads();
            int temp = in;
            in = out;
            out = in;
        }
        x[i] = buffers[i + out];
    }
}



int main()
{
    // srand(time(nullptr));

    constexpr int N = STATIC_SIZE;
    const std::vector<int> x = make_vector(N);

    int size_test = 8;
    std::vector<int> test = {3,2,5,6,8,7,4,1};
    std::vector<int> test_exclu = scan_exclu(test);
    std::cout << "Contenu du vecteur :";
    for (int value : test_exclu) {
        std::cout << " " << value;
    }
    std::cout << std::endl;
    return 0;
}
