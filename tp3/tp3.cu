#include "hip/hip_runtime.h"
#include <vector>
//#include <random>
#include <algorithm>
// #include <random>
#include <iostream>

#include "make_matrix.h"
#include <stdio.h>

#define BLOCK_SIZE 16

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

// create a row-major matrix of size (rows x cols) with random scalar numbers between -100 and +100
std::vector<float> make_matrix(int rows, int cols)
{
    std::vector<float> matrix(rows * cols);
    std::generate(matrix.begin(), matrix.end(), [](){return float(std::rand())/RAND_MAX*200.f-100.f;});
    return matrix;
}

// compute the maximal absolute difference between matrices A and B
float max_abs_diff(const std::vector<float>& A, const std::vector<float>& B)
{
    float res = 0;
    for(size_t i = 0; i < A.size(); ++i)
        res = std::max(res, std::abs(A.at(i) - B.at(i)));
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j)
__host__ __device__ int index1(int i, int j, int rows, int cols)
{
    return (j + i * cols);
}



// perform matrix multiplication C = A * B on the CPU
std::vector<float> matmul_cpu(const std::vector<float>& A, const std::vector<float>& B, int N, int M, int P)
{
    std::vector<float> res = make_matrix(N,P);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
	    int indexRes = index1(i,j,N,P);
	    res[indexRes] = 0;
            for (int k = 0; k < M; ++k) {
                int indexA = index1(i,k,N,M);
                int indexB = index1(k,j,M,P);
                res[indexRes] += A[indexA] * B[indexB];
            }
        }
    }
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j) inside sub-matrix (bi,bj)
__device__ int index2(int i, int j, int bi, int bj, int rows, int cols)
{
    int i_res = bi * BLOCK_SIZE + i;
    int j_res = bj * BLOCK_SIZE + j;
    return (j_res + i_res * cols);
}

__global__ void matmul1(float *d_A, float *d_B, float *d_C, int N, int M, int P)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < P) {
        float result = 0.0f;

        for (int k = 0; k < M; ++k) {
            float element_A = d_A[index1(i, k, N, M)];
            float element_B = d_B[index1(k, j, M, P)];
            result += element_A * element_B;
        }

        d_C[index1(i, j, N, P)] = result;
    }
}

//FAUX
__global__ void matmul2(float *d_A, float *d_B, float *d_C, int N, int M, int P)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Number of sub-matrices
    const int S = (N*P) / BLOCK_SIZE;

    // Accumulated result for one element of C
    float accu = 0.0f;

    for (int s = 0; s < S; ++s) {
        // Define shared memory for sub-matrices A and B
        __shared__ float s_A[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float s_B[BLOCK_SIZE][BLOCK_SIZE];

        // Load sub-matrices A and B into shared memory
        int row_A = by * BLOCK_SIZE + ty;
        int col_A = s * BLOCK_SIZE + tx;
        int row_B = s * BLOCK_SIZE + ty;
        int col_B = bx * BLOCK_SIZE + tx;

        if (row_A < N && col_A < M) {
            s_A[ty][tx] = d_A[row_A * M + col_A];
        } else {
            s_A[ty][tx] = 0.0f; // Padding with zeros for out-of-bounds elements
        }

        if (row_B < M && col_B < P) {
            s_B[ty][tx] = d_B[row_B * P + col_B];
        } else {
            s_B[ty][tx] = 0.0f; // Padding with zeros for out-of-bounds elements
        }

        // Synchronize threads within the block
        __syncthreads();

        // Compute partial sum in accu using s_A and s_B
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            accu += s_A[ty][k] * s_B[k][tx];
        }

        // Synchronize threads again before loading the next sub-matrices
        __syncthreads();
    }

    // Calculate the indices for the result matrix C
    int row_C = by * BLOCK_SIZE + ty;
    int col_C = bx * BLOCK_SIZE + tx;

    // Write the accumulated result to the global memory
    if (row_C < N && col_C < P) {
        d_C[row_C * P + col_C] = accu;
    }
}



int main()
{
    // srand(time(nullptr));

    const int N = 64 * BLOCK_SIZE;
    const int M = 19 * BLOCK_SIZE;
    const int P = 12 * BLOCK_SIZE;

    const dim3 threads_per_block(BLOCK_SIZE,BLOCK_SIZE,1);
    const dim3 blocks((N + BLOCK_SIZE -1)/BLOCK_SIZE, (P + BLOCK_SIZE -1)/BLOCK_SIZE, 1);

    std::vector<float> A = make_matrix(N,M);
    std::vector<float> B = make_matrix(M,P);
    std::vector<float> C = matmul_cpu(A,B,N,M,P);


    float *d_A;
    float *d_B;
    float *d_C;
    float *C_GPU = (float*) malloc(sizeof(float) * N * P);

    hipMalloc(&d_A, N * M * sizeof(float));
    hipMalloc(&d_B, M * P * sizeof(float));
    hipMalloc(&d_C, N * P * sizeof(float));
    hipMemcpy(d_A, A.data(), N * M * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B.data(), M * P * sizeof(float), hipMemcpyHostToDevice);

    matmul1<<<blocks,threads_per_block>>>(d_A,d_B,d_C, N, M, P);

    hipMemcpy(C_GPU, d_C, N * P * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> res = make_matrix(N,P);

    for (int i = 0; i < N * P; ++i) {
        res[i] = C_GPU[i];
    }

    float test = max_abs_diff(res, C);

    printf("%f\n", test);

    return 0;
}
