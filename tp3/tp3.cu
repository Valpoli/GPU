#include "hip/hip_runtime.h"
#include <vector>
//#include <random>
#include <algorithm>
// #include <random>
#include <iostream>

#include "make_matrix.h"
#include <stdio.h>

#define BLOCK_SIZE 16

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

// create a row-major matrix of size (rows x cols) with random scalar numbers between -100 and +100
std::vector<float> make_matrix(int rows, int cols)
{
    std::vector<float> matrix(rows * cols);
    std::generate(matrix.begin(), matrix.end(), [](){return float(std::rand())/RAND_MAX*200.f-100.f;});
    return matrix;
}

// compute the maximal absolute difference between matrices A and B
float max_abs_diff(const std::vector<float>& A, const std::vector<float>& B)
{
    float res = 0;
    for(size_t i = 0; i < A.size(); ++i)
        res = std::max(res, std::abs(A.at(i) - B.at(i)));
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j)
__host__ __device__ int index1(int i, int j, int rows, int cols)
{
    return (j + i * cols);
}



// perform matrix multiplication C = A * B on the CPU
std::vector<float> matmul_cpu(const std::vector<float>& A, const std::vector<float>& B, int N, int M, int P)
{
    std::vector<float> res = make_matrix(N,P);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
	    int indexRes = index1(i,j,N,P);
	    res[indexRes] = 0;
            for (int k = 0; k < M; ++k) {
                int indexA = index1(i,k,N,M);
                int indexB = index1(k,j,M,P);
                res[indexRes] += A[indexA] * B[indexB];
            }
        }
    }
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j) inside sub-matrix (bi,bj)
__device__ int index2(int i, int j, int bi, int bj, int rows, int cols)
{
    // ...
}

// __global__ matmul cpu(float *d_img, int size, int *d_hist)
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
//     if (i < N && j < M) {

//     }
// }


int main()
{
    // srand(time(nullptr));

    const int N = 64 * BLOCK_SIZE;
    const int M = 19 * BLOCK_SIZE;
    const int P = 12 * BLOCK_SIZE;

    // int threads_per_block = P;
    // int block_count = M;

    std::vector<float> testA = make_matrix(2,3);
    std::vector<float> testB = make_matrix(3,4);

    float testa[] = {2,1,4,0,1,1};
    float testb[] = {6,3,-1,0,1,1,0,4,-2,5,0,2};

    for (int i = 0; i < 6; i++)
    {
        testA[i] = testa[i];
    }
    for (int i = 0; i < 12; i++)
    {
        testB[i] = testb[i];
    }

    std::cout << "Matrix A:" << std::endl;
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 3; j++) {
            std::cout << testA[index1(i,j,2,3)] << " ";
        }
        std::cout << std::endl;
    }

    std::cout << "Matrix B:" << std::endl;
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << testB[index1(i,j,3,4)] << " ";
        }
        std::cout << std::endl;
    }

    const std::vector<float> res = matmul_cpu(testA,testB,2,3,4);

    std::cout << "res:" << std::endl;
    for (int i = 0; i < 2; i++) {
        for (int j = 0; j < 4; j++) {
            std::cout << res[index1(i,j,2,4)] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
