#include "hip/hip_runtime.h"
#include <vector>
//#include <random>
#include <algorithm>
// #include <random>
#include <iostream>

#include "make_matrix.h"
#include <stdio.h>

#define BLOCK_SIZE 16

#define CUDA_CHECK(code) { cuda_check((code), __FILE__, __LINE__); }
inline void cuda_check(hipError_t code, const char *file, int line) {
    if(code != hipSuccess) {
        fprintf(stderr,"%s:%d: [CUDA ERROR] %s\n", file, line, hipGetErrorString(code));
    }
}

// create a row-major matrix of size (rows x cols) with random scalar numbers between -100 and +100
std::vector<float> make_matrix(int rows, int cols)
{
    std::vector<float> matrix(rows * cols);
    std::generate(matrix.begin(), matrix.end(), [](){return float(std::rand())/RAND_MAX*200.f-100.f;});
    return matrix;
}

// compute the maximal absolute difference between matrices A and B
float max_abs_diff(const std::vector<float>& A, const std::vector<float>& B)
{
    float res = 0;
    for(size_t i = 0; i < A.size(); ++i)
        res = std::max(res, std::abs(A.at(i) - B.at(i)));
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j)
__host__ __device__ int index1(int i, int j, int rows, int cols)
{
    printf("%d %d %d %d\n",i,j,rows,cols);
    return (j + i * cols);
}



// perform matrix multiplication C = A * B on the CPU
std::vector<float> matmul_cpu(const std::vector<float>& A, const std::vector<float>& B, int N, int M, int P)
{
    std::vector<float> res = make_matrix(N,P);
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < P; ++j) {
	    int indexRes = index1(i,j,N,P);
	    res[indexRes] = 0;
	    printf("\n");
            for (int k = 0; k < M; ++k) {
                int indexA = index1(i,k,N,M);
                int indexB = index1(k,j,M,P);
                res[indexRes] += A[indexA] * B[indexB];
		printf("%f , %f\n",indexA, indexB);
		printf("%f * %f = %f \n",A[indexA],B[indexB],res[indexRes]);
            }
        }
    }
    return res;
}



// return the 1D index of a row-major matrix of size (rows,cols) from indices (i,j) inside sub-matrix (bi,bj)
__device__ int index2(int i, int j, int bi, int bj, int rows, int cols)
{
    // ...
}

// __global__ matmul cpu(float *d_img, int size, int *d_hist)
// {
//     int i = blockIdx.x * blockDim.x + threadIdx.x;
//     int j = blockIdx.y * blockDim.y + threadIdx.y;
//     if (i < N && j < M) {

//     }
// }


int main()
{
    // srand(time(nullptr));

    const int N = 64 * BLOCK_SIZE;
    const int M = 19 * BLOCK_SIZE;
    const int P = 12 * BLOCK_SIZE;

    // int threads_per_block = P;
    // int block_count = M;

    std::vector<float> testA = make_matrix(2,2);
    std::vector<float> testB = make_matrix(2,2);
 
    float testa[] = {1, 2, 3, 4};
    float testb[] = {5,6,0,7};

    for (int i = 0; i < 4; i++)
    {
        testA[i] = testa[i];
    }
    for (int i = 0; i < 4; i++)
    {
        testB[i] = testb[i];
    }

    const int N1 = 2;
    const int M1 = 2;
    const int P1 = 2;
    const std::vector<float> res = matmul_cpu(testA,testB,N1,M1,P1);

    printf("%f\n",testA[0]);

    std::cout << "res:" << std::endl;
    for (int i = 0; i < N1; i++) {
        for (int j = 0; j < P1; j++) {
            std::cout << res[index1(i,j,2,2)] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
